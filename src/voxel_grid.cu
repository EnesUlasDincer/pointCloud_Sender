#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <vector>

#include "../include/voxel_grid.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Eigen/Dense>

#include <thrust/device_vector.h>
#include <thrust/remove.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>

using namespace std;

// Conversion function from OBColorPoint to Point
Point toPoint(const OBColorPoint& obPoint) {
    return Point(obPoint.x, obPoint.y, obPoint.z, obPoint.r, obPoint.g, obPoint.b);
}

// Helper structures for CUDA operations
struct PointToKey {
    ui32XYZ voxel_nums;
    fXYZ voxel_lengths;
    fXYZ min_xyz;

    PointToKey(ui32XYZ _voxel_nums, fXYZ _voxel_lengths, fXYZ _min_xyz)
        : voxel_nums(_voxel_nums), voxel_lengths(_voxel_lengths), min_xyz(_min_xyz) {}

    __host__ __device__
    uint32_t operator()(const Point v) const {
        if (isnan(v.x)) return UINT32_MAX;
        uint32_t idx_x = round((v.x - min_xyz.x) / voxel_lengths.x);
        uint32_t idx_y = round((v.y - min_xyz.y) / voxel_lengths.y);
        uint32_t idx_z = round((v.z - min_xyz.z) / voxel_lengths.z);
        return idx_x + idx_y * voxel_nums.x + idx_z * voxel_nums.x * voxel_nums.y;
    }
};

struct PointToColor {
    __host__ __device__
    ui64RGB operator()(const Point p) const {
        return ui64RGB(p.r * p.r, p.g * p.g, p.b * p.b);
    }
};

struct IdxColorWeightToPoint {
    ui32XYZ voxel_nums;
    fXYZ voxel_lengths;
    fXYZ min_xyz;

    IdxColorWeightToPoint(ui32XYZ _voxel_nums, fXYZ _voxel_lengths, fXYZ _min_xyz)
        : voxel_nums(_voxel_nums), voxel_lengths(_voxel_lengths), min_xyz(_min_xyz) {}

    __host__ __device__
    Point operator()(thrust::tuple<uint32_t, ui64RGB, uint32_t> t) const {
        uint32_t idx = thrust::get<0>(t);
        ui64RGB color = thrust::get<1>(t);
        uint32_t weight = thrust::get<2>(t);

        if (voxel_nums.x == 0 || voxel_nums.y == 0 || voxel_nums.z == 0) {
            printf("Error: Invalid voxel_nums (x=%u, y=%u, z=%u)\n", voxel_nums.x, voxel_nums.y, voxel_nums.z);
            return Point();
        }

        uint32_t idx_z = idx / (voxel_nums.x * voxel_nums.y);
        uint32_t idx_y = (idx - (idx_z * voxel_nums.x * voxel_nums.y)) / voxel_nums.x;
        uint32_t idx_x = idx % voxel_nums.x;

        Point p;
        p.x = ((float) idx_x) * voxel_lengths.x + min_xyz.x;
        p.y = ((float) idx_y) * voxel_lengths.y + min_xyz.y;
        p.z = ((float) idx_z) * voxel_lengths.z + min_xyz.z;

        // printf("I am here\n");
        
        p.r = (float) round(sqrt((float) (color.r / weight)));
        p.g = (float) round(sqrt((float) (color.g / weight)));
        p.b = (float) round(sqrt((float) (color.b / weight)));
        
        // if (weight > 0) {
        //     p.r = sqrtf((float)(color.r / weight));
        //     p.g = sqrtf((float)(color.g / weight));
        //     p.b = sqrtf((float)(color.b / weight));
        // } else {
        //     p.r = p.g = p.b = 0.0f;
        // }

        return p;
    }
};


struct is_point_invalid {
    __host__ __device__
    bool operator()(const Point p) const { return isnan(p.x); }
};

struct TFAndCropPoint {
    Eigen::Matrix4f tf;
    fXYZ min_xyz;
    fXYZ max_xyz;

    TFAndCropPoint(Eigen::Matrix4f _tf, fXYZ _min_xyz, fXYZ _max_xyz)
        : tf(_tf), min_xyz(_min_xyz), max_xyz(_max_xyz) {}

    __host__ __device__
    Point operator()(const Point p) const {

        Point np;

        np.x = tf(0, 0) * p.x + tf(0, 1) * p.y + tf(0, 2) * p.z + tf(0, 3); 
        np.y = tf(1, 0) * p.x + tf(1, 1) * p.y + tf(1, 2) * p.z + tf(1, 3); 
        np.z = tf(2, 0) * p.x + tf(2, 1) * p.y + tf(2, 2) * p.z + tf(2, 3); 
        np.r = p.r;
        np.g = p.g;
        np.b = p.b;

        if (np.x < min_xyz.x || np.x > max_xyz.x ||
            np.y < min_xyz.y || np.y > max_xyz.y ||
            np.z < min_xyz.z || np.z > max_xyz.z) {
            // printf("Invalid point: (%f, %f, %f)\n", p.x, p.y, p.z);
            return Point(NAN, NAN, NAN, 0, 0, 0);
        }
    
        return np;
    }
};

uint32_t voxel_grid(thrust::device_vector<Point>& d_points, float* out, const ui32XYZ& voxel_nums, const fXYZ& voxel_lengths, const fXYZ& min_xyz) {
    uint32_t num = d_points.size();

    // Step 1: Map points to colors and voxel indices
    // std::cout << "Step 1: Map points to colors and voxel indices" << std::endl;
    thrust::device_vector<ui64RGB> d_colors(num);
    thrust::device_vector<uint32_t> d_voxel_idxs(num);
    thrust::transform(d_points.begin(), d_points.end(), d_colors.begin(), PointToColor());
    thrust::transform(d_points.begin(), d_points.end(), d_voxel_idxs.begin(), PointToKey(voxel_nums, voxel_lengths, min_xyz));

    // Step 2: Sort points by voxel index
    // std::cout << "Step 2: Sort points by voxel index" << std::endl;
    thrust::device_vector<uint32_t> d_point_idxs(num);
    thrust::sequence(d_point_idxs.begin(), d_point_idxs.end());
    thrust::sort_by_key(d_voxel_idxs.begin(), d_voxel_idxs.end(), d_point_idxs.begin());

    // Step 3: Compute voxel histogram
    // std::cout << "Step 3: Compute voxel histogram" << std::endl;
    thrust::device_vector<uint32_t> d_weights(num);
    thrust::device_vector<uint32_t> d_idx_reduced(num);
    auto new_ends = thrust::reduce_by_key(d_voxel_idxs.begin(), d_voxel_idxs.end(),
                                          thrust::constant_iterator<uint32_t>(1),
                                          d_idx_reduced.begin(), d_weights.begin());
    uint32_t num_voxels = new_ends.first - d_idx_reduced.begin();
    d_weights.resize(num_voxels);
    d_idx_reduced.resize(num_voxels);

    // Step 4: Aggregate voxel colors
    // std::cout << "Step 4: Aggregate voxel colors" << std::endl;
    thrust::device_vector<ui64RGB> d_colors_out(num_voxels);
    thrust::reduce_by_key(d_voxel_idxs.begin(), d_voxel_idxs.end(),
                          thrust::make_permutation_iterator(d_colors.begin(), d_point_idxs.begin()),
                          d_idx_reduced.begin(), d_colors_out.begin());

    // Step 5: Compute voxel centroids
    // std::cout << "Step 5: Compute voxel centroids" << std::endl;
    thrust::device_vector<Point> d_point_cloud_out(num_voxels);
    auto zip_begin = thrust::make_zip_iterator(thrust::make_tuple(d_idx_reduced.begin(), d_colors_out.begin(), d_weights.begin()));
    auto zip_end = thrust::make_zip_iterator(thrust::make_tuple(d_idx_reduced.end(), d_colors_out.end(), d_weights.end()));
    
    // std::cout << "transforming" << std::endl;
    thrust::transform(zip_begin, zip_end, d_point_cloud_out.begin(), IdxColorWeightToPoint(voxel_nums, voxel_lengths, min_xyz));

    // Copy result to output
    // std::cout << "Copying result to output" << std::endl;
    thrust::copy(d_point_cloud_out.begin(), d_point_cloud_out.end(), reinterpret_cast<Point*>(out));

    // std::cout << "Number of voxels: " << num_voxels << std::endl;
    return num_voxels;
}

#define CUDA_CHECK(call)                                               \
    do {                                                               \
        hipError_t err = call;                                        \
        if (err != hipSuccess) {                                      \
            std::cerr << "CUDA Error: " << hipGetErrorString(err)     \
                      << " at " << __FILE__ << ":" << __LINE__ << "\n";\
            exit(EXIT_FAILURE);                                        \
        }                                                              \
    } while (0)

// uint32_t transformCropAndVoxelizeCenter(std::vector<OBColorPoint>& points, float* point_cloud_out, Eigen::Matrix4f& T_camera_to_QR) {
//     size_t num_points = points.size();
//     if (num_points == 0) {
//         std::cerr << "Error: No input points\n";
//         return 0;
//     }

//     fXYZ min_xyz(-300.0f, -300.0f, -500.0f);
//     fXYZ max_xyz(+130.0f, +130.0f, 2000.0f);
//     fXYZ voxel_lengths( 1, 1, 1);
//     ui32XYZ voxel_nums(ceil((max_xyz.x - min_xyz.x) / voxel_lengths.x),
//                        ceil((max_xyz.y - min_xyz.y) / voxel_lengths.y),
//                        ceil((max_xyz.z - min_xyz.z) / voxel_lengths.z));

//     Eigen::Matrix4f tf;
//     //  = T_camera_to_QR;
//     // printf("T_camera_to_QR: \n");
//     // std::cout << T_camera_to_QR << std::endl;
//     tf << 1, 0, 0, 0,
//           0, 1, 0, 0,
//           0, 0, 1, 0,
//           0, 0, 0, 1;


//     // thrust::host_vector<Point> h_points(num_points);
//     // for (size_t i = 0; i < num_points; ++i) {
//     //     h_points[i] = toPoint(points[i]);
//     // }

//     // thrust::device_vector<Point> d_points = h_points;

//     thrust::device_vector<Point> d_points(points.size()); // Output
//     thrust::copy(
//         reinterpret_cast<const Point*>(&points[0]),
//         reinterpret_cast<const Point*>(&points[0] + points.size()),
//         d_points.begin());


//     // std
//     std::cout << "Number of points before filtering: " << d_points.size() << std::endl;

//     thrust::transform(d_points.begin(), d_points.end(), d_points.begin(), TFAndCropPoint(tf, min_xyz, max_xyz));
    
//     CUDA_CHECK(hipDeviceSynchronize());


//     size_t new_size = thrust::remove_if(d_points.begin(), d_points.end(), is_point_invalid()) - d_points.begin();
//     if (new_size > d_points.size()) {
//         std::cerr << "Error: new_size exceeds original size\n";
//         exit(EXIT_FAILURE);
//     }
//     d_points.resize(new_size);

//     CUDA_CHECK(hipDeviceSynchronize());

//     std::cout << "Number of points after filtering: " << new_size << std::endl;
//     return voxel_grid(d_points, point_cloud_out, voxel_nums, voxel_lengths, min_xyz);
// }


uint32_t transformCropAndVoxelizeCenter(OBColorPoint* points, size_t num_points, float* point_cloud_out, Eigen::Matrix4f& T_camera_to_QR) {
    if (num_points == 0) {
        std::cerr << "Error: No input points\n";
        return 0;
    }

    fXYZ min_xyz(-3000.0f, -3000.0f, -1000.0f);
    fXYZ max_xyz(+6000.0f, +6000.0f, 5000.0f);
    fXYZ voxel_lengths(1, 1, 1);
    ui32XYZ voxel_nums(
        ceil((max_xyz.x - min_xyz.x) / voxel_lengths.x),
        ceil((max_xyz.y - min_xyz.y) / voxel_lengths.y),
        ceil((max_xyz.z - min_xyz.z) / voxel_lengths.z));

    Eigen::Matrix4f tf = T_camera_to_QR;  // Use input transform
    // Eigen::Matrix4f tf;  // Use input transform
    // tf << 1, 0, 0, 0,
    //       0, 1, 0, 0,
    //       0, 0, 1, 0,
    //       0, 0, 0, 1;

    // CUDA THRUST - NO COPYING TO `std::vector`
    thrust::device_vector<Point> d_points(num_points);
    thrust::copy(
        reinterpret_cast<const Point*>(points),
        reinterpret_cast<const Point*>(points) + num_points,
        d_points.begin());

    // std::cout << "Number of points before filtering: " << d_points.size() << std::endl;

    thrust::transform(d_points.begin(), d_points.end(), d_points.begin(), TFAndCropPoint(tf, min_xyz, max_xyz));
    // CUDA_CHECK(hipDeviceSynchronize());

    size_t new_size = thrust::remove_if(d_points.begin(), d_points.end(), is_point_invalid()) - d_points.begin();
    d_points.resize(new_size);

    // CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "Number of points after filtering: " << new_size << std::endl;
    return voxel_grid(d_points, point_cloud_out, voxel_nums, voxel_lengths, min_xyz);
}
